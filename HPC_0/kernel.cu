#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "error.h"
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <iomanip>
#include <iostream>

using namespace std::chrono;

__global__ void subtractKernel(int* a, int* b, int* c, int size)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < size)
    {
        c[tid] = a[tid] - b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int* initializeIntVector(const unsigned int size)
{
    int* vector = new int[size];
    for (int i = 0; i < size; i++)
    {
        vector[i] = (rand() % 100);
    }

    return vector;
}

bool isVectorsEqual(const int* first, const int* second, int size)
{
    for (int i = 0; i < size; ++i)
    {
        if (first[i] != second[i])
        {
            return false;
        }
    }
    return true;
}

void subtractWithCUDA(const int* a, const int* b, int* c, unsigned int size)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    // Create dev variables.
    int* dev_a;
    int* dev_b;
    int* dev_c;

    // Create time events.
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate GPU buffers for three vectors (two input, one output).
    HANDLE_ERROR(hipMalloc((void**)&dev_a, size * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, size * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, size * sizeof(int)));

    // Copy input vectors from host memory to GPU buffers.
    HANDLE_ERROR(hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice));

    // Record start time.
    hipEventRecord(start, nullptr);

    // Launch a kernel on the GPU with XXX thread.
    int threadPerBlock = prop.maxThreadsPerBlock;
    int blocksPerGrid = (size + threadPerBlock - 1) / threadPerBlock;
    printf_s("CUDA launch with %d blocks and %d threads.\n", blocksPerGrid, threadPerBlock);
    subtractKernel << <blocksPerGrid, threadPerBlock >> >(dev_a, dev_b, dev_c, size);

    // hipDeviceSynchronize waits for the kernel to finish, and return any errors encountered during the launch.
    HANDLE_ERROR(hipDeviceSynchronize());

    //Calculate elapsed time.
    hipEventRecord(stop, nullptr);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf_s(">Time on GPU = %.3f milliseconds.\n", time);

    // Copy output vector from GPU buffer to host memory.
    HANDLE_ERROR(hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost));

    // Free resources.
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void subtractWithCPU(const int* a, const int* b, int* c, unsigned int size)
{
    auto start = high_resolution_clock::now();

    for (int i = 0; i < size; i++)
    {
        c[i] = a[i] - b[i];
    }

    auto end = high_resolution_clock::now();

    // Calculating total time taken by the program.
    double time_taken = duration_cast<nanoseconds>(end - start).count();
    printf_s(">Time on CPU = %.3f milliseconds.\n", time_taken *= 1e-6);
}

int main()
{
    srand(time(nullptr));
    int vectorSize = 0;

    printf_s("Input vectors size: ");
    scanf_s("%d", &vectorSize);

    const int* a = initializeIntVector(vectorSize);
    const int* b = initializeIntVector(vectorSize);
    int* cGPU = new int[vectorSize];
    int* cCPU = new int[vectorSize];

    subtractWithCUDA(a, b, cGPU, vectorSize);
    subtractWithCPU(a, b, cCPU, vectorSize);

    isVectorsEqual(cCPU, cGPU, vectorSize)
        ? printf("GPU and CPU calculations equal.\n")
        : printf("GPU and CPU calculations not equal.\n");

    return 0;
}
